#include "hip/hip_runtime.h"

#include "math.h"
#include "stdint.h"

__constant__ double box_min[2];
__constant__ double box_max[2];
// __constant__ double x[3];
__constant__ double sigt; // sigt/2
// __constant__ double2 constCont; // sqrt(weight./px) * exp(2*pi*1i*rand)
__constant__ int32_t vSize;
__constant__ int32_t lSize;
__constant__ int32_t wSize;

__constant__ double fastConstCopy[4];

__global__ void ff_single2D(double2* us, double2* wl, double2* e_l0, double* af_ang_vl,
    const double* v1, const double* v2, const double* dir_v1, const double* dir_v2)
{
    int uIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (uIdx < (vSize * wSize))
    {
        int lIdx, vIdx, wIdx;
        double bd1, bd2, d;
        double v_val_1, v_val_2;
        double v_dir_1, v_dir_2;
        double att, phase, sinptr, cosptr;
        double* x, * constCont;
        double2 vlMult, vwMult, tpath;
        double realContb, wx, wy;

        x = fastConstCopy;
        constCont = fastConstCopy + 2;

        vIdx = uIdx % vSize;
        wIdx = uIdx / vSize;

        v_val_1 = v1[vIdx];
        v_val_2 = v2[vIdx];

        v_dir_1 = dir_v1[vIdx];
        v_dir_2 = dir_v2[vIdx];

        // Cube dist
        if (v_dir_1 <= 0)
        {
            bd1 = (x[0] - box_min[0]) / abs(v_dir_1);
        }
        else
        {
            bd1 = (box_max[0] - x[0]) / abs(v_dir_1);
        }

        if (v_dir_2 <= 0)
        {
            bd2 = (x[1] - box_min[1]) / abs(v_dir_2);
        }
        else
        {
            bd2 = (box_max[1] - x[1]) / abs(v_dir_2);
        }

        d = fmin(bd1, bd2);

        // evalphaseatt
        phase = -2 * fma(v_val_1, x[0], v_val_2 * x[1]);
        att = exp(-sigt * d);
        sincospi(phase, &sinptr, &cosptr);

        // result
        for (lIdx = 0; lIdx < lSize; lIdx++)
        {
            // real mult
            realContb = att * (*(af_ang_vl + vIdx + lIdx * vSize));

            // af_ang_vl .* e_v0 .* e_l0
            vlMult.x = realContb * fma(cosptr, e_l0[lIdx].x, -sinptr * e_l0[lIdx].y);
            vlMult.y = realContb * fma(cosptr, e_l0[lIdx].y, sinptr * e_l0[lIdx].x);

            // permute(Wl,[3,2,1]) .* (af_ang_vl .* e_v0 .* e_l0)
            wx = (wl + wIdx + lIdx * wSize)->x;
            wy = (wl + wIdx + lIdx * wSize)->y;
            vwMult.x = fma(vlMult.x, wx, -vlMult.y * wy);
            vwMult.y = fma(vlMult.x, wy, vlMult.y * wx);

            // multiple with const
            tpath.x = fma(vwMult.x, constCont[0], -vwMult.y * constCont[1]);
            tpath.y = fma(vwMult.x, constCont[1], vwMult.y * constCont[0]);

            // add to us
            us[uIdx].x += tpath.x;
            us[uIdx].y += tpath.y;
        }
    }
}

int main() {
    return 0;
}