#include "hip/hip_runtime.h"

#include "math.h"
#include "stdint.h"

__constant__ double box_min[2];
__constant__ double box_max[2];
// __constant__ double x[3];
__constant__ double sigt; // sigt/2
// __constant__ double2 constCont; // sqrt(weight./px) * exp(2*pi*1i*rand)
__constant__ int32_t vSize;
__constant__ int32_t lSize;
__constant__ int32_t wSize;

__constant__ double fastConstCopy[4];

__global__ void ff_single2D(double2* us, double2* wl, double2* e_l0, double2* af_ang_vl,
    const double* v1, const double* v2, const double* dir_v1, const double* dir_v2)
{
    int uIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (uIdx < (vSize * wSize))
    {
        int lIdx, vIdx, wIdx;
        double bd1, bd2, d;
        double v_val_1, v_val_2;
        double v_dir_1, v_dir_2;
        double att, phase, sinptr, cosptr;
        double* x, * constCont;
        double2 vlMult, vwMult, tpath, attRotation, currAf_ang_vl;
        double wx, wy;

        x = fastConstCopy;
        constCont = fastConstCopy + 2;

        vIdx = uIdx % vSize;
        wIdx = uIdx / vSize;

        v_val_1 = v1[vIdx];
        v_val_2 = v2[vIdx];

        v_dir_1 = dir_v1[vIdx];
        v_dir_2 = dir_v2[vIdx];

        // Cube dist
        if (v_dir_1 <= 0)
        {
            bd1 = (x[0] - box_min[0]) / abs(v_dir_1);
        }
        else
        {
            bd1 = (box_max[0] - x[0]) / abs(v_dir_1);
        }

        if (v_dir_2 <= 0)
        {
            bd2 = (x[1] - box_min[1]) / abs(v_dir_2);
        }
        else
        {
            bd2 = (box_max[1] - x[1]) / abs(v_dir_2);
        }

        d = fmin(bd1, bd2);

        // evalphaseatt
        phase = -2 * fma(v_val_1, x[0], v_val_2 * x[1]);
        att = exp(-sigt * d);
        sincospi(phase, &sinptr, &cosptr);

        // result
        for (lIdx = 0; lIdx < lSize; lIdx++)
        {
            // attRotation - const
            currAf_ang_vl = af_ang_vl[vIdx + lIdx * vSize];
            attRotation.x = att * fma(currAf_ang_vl.x, constCont[0], -currAf_ang_vl.y * constCont[1]);
            attRotation.y = att * fma(currAf_ang_vl.x, constCont[1], currAf_ang_vl.y * constCont[0]);

            // af_ang_vl .* e_v0 .* e_l0
            vlMult.x = fma(cosptr, e_l0[lIdx].x, -sinptr * e_l0[lIdx].y);
            vlMult.y = fma(cosptr, e_l0[lIdx].y, sinptr * e_l0[lIdx].x);

            // permute(Wl,[3,2,1]) .* (af_ang_vl .* e_v0 .* e_l0)
            wx = (wl + wIdx + lIdx * wSize)->x;
            wy = (wl + wIdx + lIdx * wSize)->y;
            vwMult.x = fma(vlMult.x, wx, -vlMult.y * wy);
            vwMult.y = fma(vlMult.x, wy, vlMult.y * wx);

            // multiple with const
            tpath.x = fma(vwMult.x, attRotation.x, -vwMult.y * attRotation.y);
            tpath.y = fma(vwMult.x, attRotation.y, vwMult.y * attRotation.x);

            // add to us
            us[uIdx].x += tpath.x;
            us[uIdx].y += tpath.y;
        }
    }
}

int main() {
    return 0;
}