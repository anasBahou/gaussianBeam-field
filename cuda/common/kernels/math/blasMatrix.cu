#include "hip/hip_runtime.h"
#include "blasMatrix.h"

// compute C = A * B;
// where A in size of [n x m] and B in size of [m x k]
// A and B are stored as vectors, in such way that:
// A[0]:i_n = 0, i_m = 0. A[1]:i_n = 0, i_m = 1, ... A[m]:i_n = 1, i_m = 0

void matrixMult(hipblasHandle_t cublas_handle, ub32 n, ub32 m, ub32 k,
	const float2* A, const float2* B, float2* C)
{
    hipComplex alpha = make_hipComplex(1.0f,0.0f);
    hipComplex beta  = make_hipComplex(0.0f,0.0f);

    hipblasCgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        k, n, m, &alpha, (hipComplex*) B, k, (hipComplex*) A, m, &beta,
        (hipComplex*) C, k); 
}

void matrixMult(hipblasHandle_t cublas_handle, ub32 n, ub32 m, ub32 k,
	const double2* A, const double2* B, double2* C)
{
    hipDoubleComplex alpha = make_hipDoubleComplex(1.0,0.0);
    hipDoubleComplex beta  = make_hipDoubleComplex(0.0,0.0);

    hipblasZgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        k, n, m, &alpha, (hipDoubleComplex*) B, k, (hipDoubleComplex*) A, m, &beta,
        (hipDoubleComplex*) C, k); 
}

// compute A = A + B;
// where A and B are vectors in size of n
void matrixAdd(hipblasHandle_t cublas_handle, ub32 n,
	const float2* alpha, float2* A, const float2* B)
{
    hipblasCaxpy(cublas_handle, n, (hipComplex*) alpha,
        (hipComplex*) B, 1,(hipComplex*) A, 1);
}

void matrixAdd(hipblasHandle_t cublas_handle, ub32 n,
	const double2* alpha, double2* A, const double2* B)
{
    hipblasZaxpy(cublas_handle, n, (hipDoubleComplex*) alpha,
        (hipDoubleComplex*) B, 1,(hipDoubleComplex*) A, 1);
}
