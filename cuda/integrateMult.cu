#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "math.h"
#include "stdint.h"

__constant__ int32_t uDimProd[3];
__constant__ int32_t lDim[4];
__constant__ int32_t vDim[4];
__constant__ double2 randPhase;
__constant__ double lMixtureAlpha[32];



__forceinline __device__ double2 operator +(double2 a, double2 b)
{
    a.x = a.x + b.x;
    a.y = a.y + b.y;
    return a;
}

__forceinline __device__ double2 operator +(double2 a, double b)
{
    a.x = a.x + b;
    return a;
}

__forceinline __device__ double2 operator /(double2 a, double2 b)
{
    double2 c;
    double denominator = 1/(fma(b.x,b.x,b.y * b.y));
    c.x = (fma(a.x,b.x,a.y * b.y)) * denominator;
    c.y = (fma(a.y,b.x,-a.x * b.y)) * denominator;
    return c;
}

__forceinline __device__ double2 operator *(double a, double2 b)
{
    b.x = b.x * a;
    b.y = b.y * a;
    return b;
}

__forceinline __device__ double2 operator *(double2 a, double2 b)
{
    register double2 tmpVar = a;

    a.x = fma(a.x,b.x,- a.y * b.y);
    a.y = fma(tmpVar.x,b.y,tmpVar.y * b.x);
    return a;
}

__forceinline __device__ double2 complexSquare(double2 a)
{
    register double tmpVar = a.x;
    a.x = (a.x - a.y) * (a.x + a.y);
    a.y = 2 * tmpVar * a.y;
    return a;
}

__device__ double2 complexSqrt(double2 a)
{
    double r, absrz;
    r = hypot(a.x,a.y);
    absrz = sqrt(r) * rhypot(r + a.x,a.y);
    // absrz = sqrt(r) / (fma(sqrt((r + a.x)),(r + a.x),a.y * a.y));
    a.x = absrz * (a.x + r);
    a.y = absrz * a.y;
    return a;
}

__forceinline __device__ double2 complexExponent(double2 a)
{
    register double expr,sina,cosa;
    expr = exp(a.x);
    sincos(a.y, &sina, &cosa);
    a.x = expr * cosa;
    a.y = expr * sina;
    return a;
}

__forceinline __device__ void complexIncrease(double2* address, double2 val)
{
    //atomicAdd(&(*address).x, val.x);
    //atomicAdd(&(*address).y, val.y);
    (*address).x += val.x;
    (*address).y += val.y;
}

__global__ void integrateMult(double2* u,
    const double2* lMixtureMu1, const double2* lMixtureMu2, const double2* lMixtureMu3, const double2* lMixtureC,
    const double2* vMixtureMu1, const double2* vMixtureMu2, const double2* vMixtureMu3, const double2* vMixtureC)
{
    int uIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (uIdx < uDimProd[2])
    {
        int3 uDimIdx;
        uDimIdx.z = uIdx / (uDimProd[1]);
        uDimIdx.y = (uIdx - uDimIdx.z * (uDimProd[1])) / uDimProd[0];
        uDimIdx.x = uIdx % uDimProd[0];
        int lIdx = lDim[0] * (lDim[1] > 1)* uDimIdx.x + lDim[0] * lDim[1] * (lDim[2] > 1)* uDimIdx.y + lDim[0] * lDim[1] * lDim[2] * (lDim[3] > 1)* uDimIdx.z;
        int vIdx = (vDim[1] > 1)* uDimIdx.x + vDim[1] * (vDim[2] > 1)* uDimIdx.y + vDim[1] * vDim[2] * (vDim[3] > 1)* uDimIdx.z;
        double2 sqrtMu;
        int currlIdx;
        double2 vMu1 = vMixtureMu1[vIdx];
        double2 vMu2 = vMixtureMu2[vIdx];
        double2 vMu3 = vMixtureMu3[vIdx];
        double2 vC = vMixtureC[vIdx];

        for (int mixtureIdx = 0; mixtureIdx < lDim[0]; mixtureIdx++)
        {
            currlIdx = lIdx + mixtureIdx;
            sqrtMu = complexSqrt(complexSquare(lMixtureMu1[currlIdx] + vMu1) +
                complexSquare(lMixtureMu2[currlIdx] + vMu2) +
                complexSquare(lMixtureMu3[currlIdx] + vMu3));
            complexIncrease(u + uIdx, (lMixtureAlpha[mixtureIdx] * randPhase) * (complexExponent(lMixtureC[currlIdx] + vC + sqrtMu) / sqrtMu));
            // u[uIdx] = u[uIdx] + (lMixtureAlpha[mixtureIdx] * randPhase) * (complexExponent(lMixtureC[currlIdx] + vC + sqrtMu) / sqrtMu);
        }
    }
}

int main() {
    return 0;

}